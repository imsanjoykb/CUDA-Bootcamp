#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"
#define N 300000
__global__ void kernel_1()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_2()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_3()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_4()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
int main()
{
    int n_stream=5;
    hipStream_t *stream=(hipStream_t*)malloc(n_stream*sizeof(hipStream_t));
    for(int i=0;i<n_stream;i++)
    {
        hipStreamCreate(&stream[i]);
    }
    dim3 block(1);
    dim3 grid(1);
    hipEvent_t start,stop;
    hipEvent_t * event=(hipEvent_t *)malloc(n_stream*sizeof(hipEvent_t));
    for(int i=0;i<n_stream;i++)
    {
        hipEventCreateWithFlags(&event[i],hipEventDisableTiming);
    }
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for(int i=0;i<n_stream;i++)
    {
        kernel_1<<<grid,block,0,stream[i]>>>();
        kernel_2<<<grid,block,0,stream[i]>>>();
        kernel_3<<<grid,block,0,stream[i]>>>();
        kernel_4<<<grid,block,0,stream[i]>>>();
        hipEventRecord(event[i],stream[i]);
        hipStreamWaitEvent(stream[n_stream-1],event[i],0);
    }
    hipEventRecord(stop);
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time,start,stop);

    for(int i=0;i<n_stream;i++)
    {
        hipStreamDestroy(stream[i]);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    for(int i=0;i<n_stream;i++)
    {
        hipEventDestroy(event[i]);
    }
    free(stream);
    free(event);
    return 0;
}
