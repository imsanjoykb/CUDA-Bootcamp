#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc,char** argv)
{
    printf("%s Starting ...\n",argv[0]);
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if(error_id!=hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n ->%s\n",
              (int)error_id,hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
    if(deviceCount==0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n",deviceCount);
    }
    int dev=0,driverVersion=0,runtimeVersion=0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);
    printf("Device %d:\"%s\"\n",dev,deviceProp.name);
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version         %d.%d  /  %d.%d\n",
        driverVersion/1000,(driverVersion%100)/10,
        runtimeVersion/1000,(runtimeVersion%100)/10);
    printf("  CUDA Capability Major/Minor version number:   %d.%d\n",
        deviceProp.major,deviceProp.minor);
    printf("  Total amount of global memory:                %.2f GBytes (%llu bytes)\n",
            (float)deviceProp.totalGlobalMem/pow(1024.0,3),deviceProp.totalGlobalMem);
    printf("  GPU Clock rate:                               %.0f MHz (%0.2f GHz)\n",
            deviceProp.clockRate*1e-3f,deviceProp.clockRate*1e-6f);
    printf("  Memory Bus width:                             %d-bits\n",
            deviceProp.memoryBusWidth);
    if (deviceProp.l2CacheSize)
    {
        printf("  L2 Cache Size:                            	%d bytes\n",
                deviceProp.l2CacheSize);
    }
    printf("  Max Texture Dimension Size (x,y,z)            1D=(%d),2D=(%d,%d),3D=(%d,%d,%d)\n",
            deviceProp.maxTexture1D,deviceProp.maxTexture2D[0],deviceProp.maxTexture2D[1]
            ,deviceProp.maxTexture3D[0],deviceProp.maxTexture3D[1],deviceProp.maxTexture3D[2]);
    printf("  Max Layered Texture Size (dim) x layers       1D=(%d) x %d,2D=(%d,%d) x %d\n",
            deviceProp.maxTexture1DLayered[0],deviceProp.maxTexture1DLayered[1],
            deviceProp.maxTexture2DLayered[0],deviceProp.maxTexture2DLayered[1],
            deviceProp.maxTexture2DLayered[2]);
    printf("  Total amount of constant memory               %lu bytes\n",
            deviceProp.totalConstMem);
    printf("  Total amount of shared memory per block:      %lu bytes\n",
            deviceProp.sharedMemPerBlock);
    printf("  Total number of registers available per block:%d\n",
            deviceProp.regsPerBlock);
    printf("  Wrap size:                                    %d\n",deviceProp.warpSize);
    printf("  Maximun number of thread per multiprocesser:  %d\n",
            deviceProp.maxThreadsPerMultiProcessor);
    printf("  Maximun number of thread per block:           %d\n",
            deviceProp.maxThreadsPerBlock);
    printf("  Maximun size of each dimension of a block:    %d x %d x %d\n",
            deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
    printf("  Maximun size of each dimension of a grid:     %d x %d x %d\n",
            deviceProp.maxGridSize[0],
	    deviceProp.maxGridSize[1],
	    deviceProp.maxGridSize[2]);
    printf("  Maximu memory pitch                           %lu bytes\n",deviceProp.memPitch);
    printf("----------------------------------------------------------\n");
    printf("Number of multiprocessors:                      %d\n", deviceProp.multiProcessorCount);
    printf("Total amount of constant memory:                %4.2f KB\n",
	deviceProp.totalConstMem/1024.0);
    printf("Total amount of shared memory per block:        %4.2f KB\n",
     deviceProp.sharedMemPerBlock/1024.0);
    printf("Total number of registers available per block:  %d\n",
    deviceProp.regsPerBlock);
    printf("Warp size                                       %d\n", deviceProp.warpSize);
    printf("Maximum number of threads per block:            %d\n", deviceProp.maxThreadsPerBlock);
    printf("Maximum number of threads per multiprocessor:  %d\n",
	deviceProp.maxThreadsPerMultiProcessor);
    printf("Maximum number of warps per multiprocessor:     %d\n",
	deviceProp.maxThreadsPerMultiProcessor/32);
    return EXIT_SUCCESS;
   
}
