
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void hello_world(void)
{
  printf("GPU: Hello world!\n");
}
int main(int argc,char **argv)
{
  printf("CPU: Hello world!\n");
  hello_world<<<1,10>>>();
  hipDeviceReset();//if no this line ,it can not output hello world from gpu
  return 0;
}
