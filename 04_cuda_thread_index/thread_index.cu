#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"

__global__ void printThreadIndex(float *A,const int nx,const int ny)
{
  int ix=threadIdx.x+blockIdx.x*blockDim.x;
  int iy=threadIdx.y+blockIdx.y*blockDim.y;
  unsigned int idx=iy*nx+ix;
  printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d)"
          "global index %2d ival %2d\n",threadIdx.x,threadIdx.y,
          blockIdx.x,blockIdx.y,ix,iy,idx,A[idx]);
}
int main(int argc,char** argv)
{
  initDevice(0);
  int nx=8,ny=6;
  int nxy=nx*ny;
  int nBytes=nxy*sizeof(float);

  //Malloc
  float* A_host=(float*)malloc(nBytes);
  initialData(A_host,nxy);
  printMatrix(A_host,nx,ny);

  //hipMalloc
  float *A_dev=NULL;
  CHECK(hipMalloc((void**)&A_dev,nBytes));

  hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice);

  dim3 block(4,2);
  dim3 grid((nx-1)/block.x+1,(ny-1)/block.y+1);

  printThreadIndex<<<grid,block>>>(A_dev,nx,ny);

  CHECK(hipDeviceSynchronize());
  hipFree(A_dev);
  free(A_host);

  hipDeviceReset();
  return 0;
}
