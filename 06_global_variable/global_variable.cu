#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ float devData;
__global__ void checkGlobalVariable()
{
    printf("Device: The value of the global variable is %f\n",devData);
    devData+=2.0;
}
int main()
{
    float value=3.14f;
    hipMemcpyToSymbol(HIP_SYMBOL(devData),&value,sizeof(float));
    printf("Host: copy %f to the global variable\n",value);
    checkGlobalVariable<<<1,1>>>();
    hipMemcpyFromSymbol(&value,HIP_SYMBOL(devData),sizeof(float));
    printf("Host: the value changed by the kernel to %f \n",value);
    hipDeviceReset();
    return EXIT_SUCCESS;
}
