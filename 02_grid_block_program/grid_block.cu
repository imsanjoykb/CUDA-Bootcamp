#include <hip/hip_runtime.h>
#include <stdio.h>
int main(int argc,char ** argv)
{
  int nElem=1024;
  dim3 block(1024);
  dim3 grid((nElem-1)/block.x+1);
  printf("grid.x %d block.x %d\n",grid.x,block.x);

  block.x=512;
  grid.x=(nElem-1)/block.x+1;
  printf("grid.x %d block.x %d\n",grid.x,block.x);

  block.x=256;
  grid.x=(nElem-1)/block.x+1;
  printf("grid.x %d block.x %d\n",grid.x,block.x);

  block.x=128;
  grid.x=(nElem-1)/block.x+1;
  printf("grid.x %d block.x %d\n",grid.x,block.x);

  hipDeviceReset();
  return 0;
}
