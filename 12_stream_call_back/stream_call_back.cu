#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"
#define N_REPEAT 10
#define N_SEGMENT 16
void CUDART_CB my_callback(hipStream_t stream,hipError_t status,void * data)
{
    printf("call back from stream:%d\n",*((int *)data));
}
void sumArrays(float * a,float * b,float * res,const int size)
{
    for(int i=0;i<size;i+=4)
    {
        res[i]=a[i]+b[i];
        res[i+1]=a[i+1]+b[i+1];
        res[i+2]=a[i+2]+b[i+2];
        res[i+3]=a[i+3]+b[i+3];
    }
}
__global__ void sumArraysGPU(float*a,float*b,float*res,int N)
{
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < N)
    //for delay
    {
        for(int j=0;j<N_REPEAT;j++)
            res[idx]=a[idx]+b[idx];
    }

}
int main(int argc,char **argv)
{
    // set up device
    initDevice(0);
    double iStart,iElaps;
    iStart=cpuSecond();
    int nElem=1<<24;
    printf("Vector size:%d\n",nElem);
    int nByte=sizeof(float)*nElem;
    float * a_h,*b_h,*res_h,*res_from_gpu_h;
    CHECK(hipHostAlloc((float**)&a_h,nByte,hipHostMallocDefault));
    CHECK(hipHostAlloc((float**)&b_h,nByte,hipHostMallocDefault));
    CHECK(hipHostAlloc((float**)&res_h,nByte,hipHostMallocDefault));
    CHECK(hipHostAlloc((float**)&res_from_gpu_h,nByte,hipHostMallocDefault));

    hipMemset(res_h,0,nByte);
    hipMemset(res_from_gpu_h,0,nByte);

    float *a_d,*b_d,*res_d;
    CHECK(hipMalloc((float**)&a_d,nByte));
    CHECK(hipMalloc((float**)&b_d,nByte));
    CHECK(hipMalloc((float**)&res_d,nByte));

    initialData(a_h,nElem);
    initialData(b_h,nElem);

    sumArrays(a_h,b_h,res_h,nElem);
    dim3 block(512);
    dim3 grid((nElem-1)/block.x+1);


    //asynchronous calculation
    int iElem=nElem/N_SEGMENT;
    hipStream_t stream[N_SEGMENT];
    for(int i=0;i<N_SEGMENT;i++)
    {
        CHECK(hipStreamCreate(&stream[i]));
    }
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    for(int i=0;i<N_SEGMENT;i++)
    {
        int ioffset=i*iElem;
        CHECK(hipMemcpyAsync(&a_d[ioffset],&a_h[ioffset],nByte/N_SEGMENT,hipMemcpyHostToDevice,stream[i]));
        CHECK(hipMemcpyAsync(&b_d[ioffset],&b_h[ioffset],nByte/N_SEGMENT,hipMemcpyHostToDevice,stream[i]));
        sumArraysGPU<<<grid,block,0,stream[i]>>>(&a_d[ioffset],&b_d[ioffset],&res_d[ioffset],iElem);
        CHECK(hipMemcpyAsync(&res_from_gpu_h[ioffset],&res_d[ioffset],nByte/N_SEGMENT,hipMemcpyDeviceToHost,stream[i]));
        CHECK(hipStreamAddCallback(stream[i],my_callback,(void *)(stream+i),0));
    }
    //timer
    CHECK(hipEventRecord(stop, 0));
    int counter=0;
    while (hipEventQuery(stop)==hipErrorNotReady)
    {
        counter++;
    }
    printf("cpu counter:%d\n",counter);
    iElaps=cpuSecond()-iStart;
    printf("Asynchronous Execution configuration<<<%d,%d>>> Time elapsed %f sec\n",grid.x,block.x,iElaps);
    checkResult(res_h,res_from_gpu_h,nElem);
    for(int i=0;i<N_SEGMENT;i++)
    {
        CHECK(hipStreamDestroy(stream[i]));
    }
    hipFree(a_d);
    hipFree(b_d);
    hipFree(a_h);
    hipFree(b_h);
    hipFree(res_h);
    hipFree(res_from_gpu_h);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
