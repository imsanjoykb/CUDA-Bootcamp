#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"
void sumMatrix2D_CPU(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
  float * a=MatA;
  float * b=MatB;
  float * c=MatC;
  for(int j=0;j<ny;j++)
  {
    for(int i=0;i<nx;i++)
    {
      c[i]=a[i]+b[i];
    }
    c+=nx;
    b+=nx;
    a+=nx;
  }
}
__global__ void sumMatrix(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx=ix+iy*ny;
    if (ix<nx && iy<ny)
    {
      MatC[idx]=MatA[idx]+MatB[idx];
    }
}

int main(int argc,char** argv)
{
  printf("strating...\n");
  initDevice(0);
  int nx=1<<12;
  int ny=1<<12;
  int nxy=nx*ny;
  int nBytes=nxy*sizeof(float);

  //Malloc
  float* A_host=(float*)malloc(nBytes);
  float* B_host=(float*)malloc(nBytes);
  float* C_host=(float*)malloc(nBytes);
  float* C_from_gpu=(float*)malloc(nBytes);
  initialData(A_host,nxy);
  initialData(B_host,nxy);

  //hipMalloc
  float *A_dev=NULL;
  float *B_dev=NULL;
  float *C_dev=NULL;
  CHECK(hipMalloc((void**)&A_dev,nBytes));
  CHECK(hipMalloc((void**)&B_dev,nBytes));
  CHECK(hipMalloc((void**)&C_dev,nBytes));


  CHECK(hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_dev,B_host,nBytes,hipMemcpyHostToDevice));

  int dimx=32;
  int dimy=32;

  // cpu compute
  hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost);
  double iStart=cpuSecond();
  sumMatrix2D_CPU(A_host,B_host,C_host,nx,ny);
  double iElaps=cpuSecond()-iStart;
  printf("CPU Execution Time elapsed %f sec\n",iElaps);

  // 2d block and 2d grid
  dim3 block_0(dimx,dimy);
  dim3 grid_0((nx-1)/block_0.x+1,(ny-1)/block_0.y+1);
  iStart=cpuSecond();
  sumMatrix<<<grid_0,block_0>>>(A_dev,B_dev,C_dev,nx,ny);
  CHECK(hipDeviceSynchronize());
  iElaps=cpuSecond()-iStart;
  printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n",
        grid_0.x,grid_0.y,block_0.x,block_0.y,iElaps);
  CHECK(hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost));
  checkResult(C_host,C_from_gpu,nxy);
  
  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  free(A_host);
  free(B_host);
  free(C_host);
  free(C_from_gpu);
  hipDeviceReset();
  return 0;
}
