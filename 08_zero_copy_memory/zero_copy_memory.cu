#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"


void sumArrays(float * a,float * b,float * res,const int size)
{
  for(int i=0;i<size;i+=4)
  {
    res[i]=a[i]+b[i];
    res[i+1]=a[i+1]+b[i+1];
    res[i+2]=a[i+2]+b[i+2];
    res[i+3]=a[i+3]+b[i+3];
  }
}
__global__ void sumArraysGPU(float*a,float*b,float*res)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  res[i]=a[i]+b[i];
}
int main(int argc,char **argv)
{
  int dev = 0;
  hipSetDevice(dev);
  int power=10;
  if(argc>=2)
    power=atoi(argv[1]);
  int nElem=1<<power;
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  float *res_from_gpu_h=(float*)malloc(nByte);
  float *res_h=(float*)malloc(nByte);
  memset(res_h,0,nByte);
  memset(res_from_gpu_h,0,nByte);

  float *a_host,*b_host,*res_d;
  double iStart,iElaps;
  dim3 block(1024);
  dim3 grid(nElem/block.x);
  res_from_gpu_h=(float*)malloc(nByte);
  float *a_dev,*b_dev;
  CHECK(hipHostAlloc((float**)&a_host,nByte,hipHostMallocMapped));
  CHECK(hipHostAlloc((float**)&b_host,nByte,hipHostMallocMapped));
  CHECK(hipMalloc((float**)&res_d,nByte));
  initialData(a_host,nElem);
  initialData(b_host,nElem);

 //=============================================================//
  iStart = cpuSecond();
  CHECK(hipHostGetDevicePointer((void**)&a_dev,(void*) a_host,0));
  CHECK(hipHostGetDevicePointer((void**)&b_dev,(void*) b_host,0));
  sumArraysGPU<<<grid,block>>>(a_dev,b_dev,res_d);
  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
  iElaps = cpuSecond() - iStart;
 //=============================================================//
  printf("zero copy memory elapsed %lf ms \n", iElaps);
  printf("Execution configuration<<<%d,%d>>>\n",grid.x,block.x);
//-----------------------normal memory---------------------------
  float *a_h_n=(float*)malloc(nByte);
  float *b_h_n=(float*)malloc(nByte);
  float *res_h_n=(float*)malloc(nByte);
  float *res_from_gpu_h_n=(float*)malloc(nByte);
  memset(res_h_n,0,nByte);
  memset(res_from_gpu_h_n,0,nByte);

  float *a_d_n,*b_d_n,*res_d_n;
  CHECK(hipMalloc((float**)&a_d_n,nByte));
  CHECK(hipMalloc((float**)&b_d_n,nByte));
  CHECK(hipMalloc((float**)&res_d_n,nByte));

  initialData(a_h_n,nElem);
  initialData(b_h_n,nElem);
//=============================================================//
  iStart = cpuSecond();
  CHECK(hipMemcpy(a_d_n,a_h_n,nByte,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(b_d_n,b_h_n,nByte,hipMemcpyHostToDevice));
  sumArraysGPU<<<grid,block>>>(a_d_n,b_d_n,res_d_n);
  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
  iElaps = cpuSecond() - iStart;
//=============================================================//
  printf("device memory elapsed %lf ms \n", iElaps);
  printf("Execution configuration<<<%d,%d>>>\n",grid.x,block.x);
//--------------------------------------------------------------------

  sumArrays(a_host,b_host,res_h,nElem);
  checkResult(res_h,res_from_gpu_h,nElem);

  hipHostFree(a_host);
  hipHostFree(b_host);
  hipFree(res_d);
  free(res_h);
  free(res_from_gpu_h);

  hipFree(a_d_n);
  hipFree(b_d_n);
  hipFree(res_d_n);

  free(a_h_n);
  free(b_h_n);
  free(res_h_n);
  free(res_from_gpu_h_n);
  return 0;
}
